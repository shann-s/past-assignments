#include "hip/hip_runtime.h"
// Simple Matrix Multiply - Workshop 6
// w6.cu

#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <chrono>
// add CUDA runtime header file
#include <hip/hip_runtime.h>
#include ""
using namespace std::chrono;

const int ntpb = 32; // number of threads per block

					 // - add your kernel here
__global__ void initialize(float* a, float* b, float* c, int n) {

	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int row = blockIdx.y*blockDim.y + threadIdx.y;

	if (row < n && col < n) {
		float sum = 0.0f;
		for (int i = 0; i < n; i++) {
			sum += a[col * n + i] * b[i * n + row];
		}
		c[col * n + row] = sum;
	}
}





					 // check reports error if any
					 //
void check(const char* msg, const hipError_t err) {
	if (err != hipSuccess)
		std::cerr << "*** " << msg << ":" << hipGetErrorString(err) << " ***\n";
}

// display matrix M, which is stored in row-major order
//
void display(const char* str, const float* M, int nr, int nc)
{
	std::cout << str << std::endl;
	std::cout << std::fixed << std::setprecision(4);
	for (int i = 0; i < nr; i++) {
		for (int j = 0; j < nc; j++)
			std::cout << std::setw(10)
			<< M[i * nc + j];
		std::cout << std::endl;
	}
	std::cout << std::endl;
}

// report system time
//
void reportTime(const char* msg, steady_clock::duration span) {
	auto ms = duration_cast<milliseconds>(span);
	std::cout << msg << " - took - " <<
		ms.count() << " millisecs" << std::endl;
}

// matrix multiply
//
void sgemm(float* h_a, float* h_b, float* h_c, int n) {

	// - calculate number of blocks for n rows
	int blocks = (n + ntpb - 1) / ntpb;
	// allocate memory for matrices d_a, d_b, d_c on the device

	// - add your allocation code here
	float* d_a = nullptr;
	float* d_b = nullptr;
	float* d_c = nullptr;
	hipMalloc((void**)&d_a, n * n * sizeof(float));
	hipMalloc((void**)&d_b, n * n * sizeof(float));
	hipMalloc((void**)&d_c, n * n * sizeof(float));
	// copy h_a and h_b to d_a and d_b (host to device)
	// - add your copy code here
	hipMemcpy(d_a, h_a, n * n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, n * n * sizeof(float), hipMemcpyHostToDevice);


	// launch execution configuration
	// - define your 2D grid of blocks
	dim3 dGrid(blocks, blocks, 1);
	// - define your 2D block of threads
	dim3 dBlock(ntpb, ntpb, 1);
	// - launch your execution configuration
	hipError_t error = hipGetLastError();
	initialize << <dGrid, dBlock >> > (d_a, d_b, d_c, n);
	// - check for launch termination
	check("Launch Termination", error);
	// copy d_c to h_c (device to host)
	// - add your copy code here
	hipMemcpy(h_c, d_c, n * n * sizeof(float), hipMemcpyDeviceToHost);

	// deallocate device memory
	// - add your deallocation code here
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	// reset the device
	hipDeviceReset();
}

int main(int argc, char* argv[]) {
	if (argc != 2) {
		std::cerr << argv[0] << ": invalid number of arguments\n";
		std::cerr << "Usage: " << argv[0] << "  size_of_vector\n";
		return 1;
	}
	int n = std::atoi(argv[1]); // number of rows/columns in h_a, h_b, h_c 
	steady_clock::time_point ts, te;

	// allocate host memory
	ts = steady_clock::now();
	float* h_a = new float[n * n];
	float* h_b = new float[n * n];
	float* h_c = new float[n * n];

	// populate host matrices a and b
	for (int i = 0, kk = 0; i < n; i++)
		for (int j = 0; j < n; j++, kk++)
			h_a[kk] = h_b[kk] = (float)kk / (n * n);
	te = steady_clock::now();
	reportTime("allocation and initialization", te - ts);

	// h_c = h_a * h_b
	ts = steady_clock::now();
	sgemm(h_a, h_b, h_c, n);
	te = steady_clock::now();
	reportTime("matrix-matrix multiplication", te - ts);

	// display results
	if (n <= 5) {
		display("h_a :", h_a, n, n);
		display("h_b :", h_b, n, n);
		display("h_c = h_a h_b :", h_c, n, n);
	}

	// check correctness
	std::cout << "correctness test ..." << std::endl;
	for (int i = 0; i < n; i++)
		for (int j = 0; j < n; j++) {
			float sum = 0.0f;
			for (int k = 0; k < n; k++)
				sum += h_a[i * n + k] * h_b[k * n + j];
			if (std::abs(h_c[i * n + j] - sum) > 1.0e-3f)
				std::cout << "[" << i << "," << j << "]" << h_c[i * n + j]
				<< " != " << sum << std::endl;
		}
	std::cout << "done" << std::endl;

	// deallocate host memory
	delete[] h_a;
	delete[] h_b;
	delete[] h_c;
}