#include "hip/hip_runtime.h"
// Initialize Memory using a Kernel - Two-Dimensional Data
// matrix_thread_id.cu

#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>

const unsigned ntpb = 32;

__global__ void initialize(float* a, float v, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < n && j < n)
		a[j * n + i] = v;
}

int main(int argc, char* argv[]) {
	if (argc != 3) {
		std::cerr << "***Incorrect number of arguments***\n";
		return 1;
	}
	unsigned n = atoi(argv[1]);
	float v = atof(argv[2]);

	int nb = (n + ntpb - 1) / ntpb;
	std::cout << "n = " << n << ", No of Blocks = " << nb
		<< ", No of Threads Per Block = " << ntpb << std::endl;

	float* d_a = nullptr;
	hipMalloc((void**)&d_a, n * n * sizeof(float));
	if (!d_a) {
		std::cerr << "***Out of Memory***\n";
		return 2;
	}
	float* h_a = new float[n * n];

	// launch
	dim3 dGrid(nb, nb, 1);
	dim3 dBlock(ntpb, ntpb, 1);
	initialize << <dGrid, dBlock >> >(d_a, v, n);

	// copy from device to host memory
	hipMemcpy(h_a, d_a, n * n * sizeof(float), hipMemcpyDeviceToHost);

	// check correctness
	for (int i = 0; i < n * n; i++)
		if (h_a[i] != v) std::cout << h_a[i] << "" << v << std::endl;
	std::cout << "done" << std::endl;

	hipFree(d_a);
	delete[] h_a;
	hipDeviceReset();
}