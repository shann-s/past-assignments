#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <cstdlib>

#define BLOCKS 128
#define THREADS 32
#define PI 3.1415926535  // known value of pi

__global__ void gpu_monte_carlo(float *estimate, hiprandState *states, float n) {
	unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
	float points_in_circle = 0;
	float x, y;

	hiprand_init(1234, tid, 0, &states[tid]);  // 	Initialize CURAND


	for (int i = 0; i < n; i++) {
		x = hiprand_uniform(&states[tid]);
		y = hiprand_uniform(&states[tid]);
		points_in_circle += (x*x + y*y <= 1.0f); // count if x & y is in the circle.
	}
	estimate[tid] = 4.0f * points_in_circle / n; // return estimate of pi
}

float host_monte_carlo(long trials) {
	float x, y;
	long points_in_circle = 0;
	for (long i = 0; i < trials; i++) {
		x = rand() / (float)RAND_MAX;
		y = rand() / (float)RAND_MAX;
		points_in_circle += (x*x + y*y <= 1.0f);
	}
	return 4.0f * points_in_circle / trials;
}

int main(int argc, char** argv) {
	clock_t start, stop;
	hiprandState *devStates;
	float n = std::atoi(argv[1]);
	dim3 dimGrid(BLOCKS, 1, 1);  // Grid dimensions
	dim3 dimBlock(THREADS, 1, 1);  // Block dimensions
	float *dev, *host;
	size_t size = BLOCKS * THREADS * sizeof(float);  //Array memory size
	printf("# of trials per thread = %.0f, # of blocks = %d, # of threads/block = %d.\n", n,
		BLOCKS, THREADS);

	start = clock();
	host = (float *)malloc(size);  //  Allocate array on host
	hipMalloc((void **)&dev, size);  // Allocate array on device
	hipMemset(dev, 0, size);
	hipMalloc((void **)&devStates, THREADS * BLOCKS * sizeof(hiprandState));
	gpu_monte_carlo << <BLOCKS, THREADS >> >(dev, devStates, n);
	hipMemcpy(host, dev, size, hipMemcpyDeviceToHost); // return results 

	float pi_gpu = 0.0f;
	for (int i = 0; i < BLOCKS * THREADS; i++) {
		pi_gpu += host[i];
	}

	pi_gpu /= (BLOCKS * THREADS);

	stop = clock();

	printf("GPU pi calculated in %f s.\n", (stop - start) / (float)CLOCKS_PER_SEC);

	start = clock();
	//float pi_cpu = host_monte_carlo(BLOCKS * THREADS * n);
	stop = clock();
	printf("CPU pi calculated in %f s.\n", (stop - start) / (float)CLOCKS_PER_SEC);

	printf("CUDA estimate of PI = %f [error of %f]\n", pi_gpu, pi_gpu - PI);
	//printf("CPU estimate of PI = %f [error of %f]\n", pi_cpu, pi_cpu - PI);

	return 0;
}