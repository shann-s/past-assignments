#include "hip/hip_runtime.h"
// Vector Magnitude - Workshop 5
// w5.cu

#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <chrono>
#include <cstdlib>
#include <algorithm>
// insert thrust header files here
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/transform_reduce.h>
#include <thrust/iterator/constant_iterator.h>
using namespace std::chrono;

// report system time
//
void reportTime(const char* msg, steady_clock::duration span) {
	auto ms = duration_cast<milliseconds>(span);
	std::cout << msg << " - took - " <<
		ms.count() << " millisecs" << std::endl;
}

// Square Function Object - add class definition here

class Square {
	const int inc;
public:

	int operator()(int x) const { return sqrt(x);  }

};



// magnitude - add calculation steps here
//
float magnitude(thrust::host_vector<int> &V) {
	float result;

	// ... add Thrust calls
	thrust::device_vector<int> d_a = V;
	Square(d_a);

	return result;
}

int main(int argc, char** argv) {
	if (argc != 2) {
		std::cerr << argv[0] << ": invalid number of arguments\n";
		std::cerr << "Usage: " << argv[0] << "  size_of_vector\n";
		return 1;
	}
	int n = std::atoi(argv[1]); // number of elements
	steady_clock::time_point ts, te;

	// Thrust definition of host vector
	thrust::host_vector<int> V(n);

	// initialize the host vector
	ts = steady_clock::now();
	std::generate(V.begin(), V.end(), std::rand);
	te = steady_clock::now();
	reportTime("initialization", te - ts);

	// calculate the magnitude of the host vector
	ts = steady_clock::now();
	float len = magnitude(V);
	te = steady_clock::now();
	reportTime("magnitude calculation", te - ts);

	// display the magnitude
	std::cout << std::fixed << std::setprecision(4);
	std::cout << "Magnitude : " << len << std::endl;
}