#include "hip/hip_runtime.h"
// Reduction - Workshop 7
// w7_1.cu

#include <iostream>
#include <cstdlib>
#include <ctime>
// CUDA header file
#include <hip/hip_runtime.h>
#include ""

void init(float* a, int n) {
	float f = 1.0f / RAND_MAX;
	for (int i = 0; i < n; i++)
		a[i] = std::rand() * f; // [0.0f 1.0f]
}

// kernel code
__global__ void reduction(float* a, float* b, int n) {

	float sum = 0.0f;
	for (int i = 0; i < n; i++) 
		a[0] += a[i] * b[i];
}








int main(int argc, char** argv) {
	// interpret command-line arguments
	if (argc != 2) {
		std::cerr << argv[0] << ": invalid number of arguments\n";
		std::cerr << "Usage: " << argv[0] << "  size_of_vectors\n";
		return 1;
	}
	int n = std::atoi(argv[1]);
	std::srand((unsigned)time(nullptr));

	// host vectors
	float* h_a = new float[n];
	float* h_b = new float[n];
	init(h_a, n);
	init(h_b, n);
	// dot product on the host
	float h_h = 0.f;
	for (int i = 0; i < n; i++)
		h_h += h_a[i] * h_b[i];

	// allocate memory for device vectors (d_a[n], d_b[n])
	float* d_a = nullptr;
	float* d_b = nullptr; 
	hipMalloc((void**)&d_a, n * sizeof(float));
	hipMalloc((void**)&d_b, n * sizeof(float));

	// copy host vectors to device vectors h_a -> d_a, h_b -> d_b
	hipMemcpy(d_a, h_a, n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, n * sizeof(float), hipMemcpyHostToDevice);

	// launch the grid of threads
	reduction << <1, n >> > (d_a, d_b, n);

	// copy the result from the device to the host d_a -> h_c
	float* h_c = nullptr;
	hipMemcpy(h_c, d_a, n * sizeof(float), hipMemcpyDeviceToHost);


	// compare the results
	std::cout << "Device = " << h_c << "\nHost   = " << h_h << std::endl;

	// free device memory
	hipFree(d_a);
	hipFree(d_b);


	// free host memory
	delete[] h_a;
	delete[] h_b;
	delete[] h_c;
}